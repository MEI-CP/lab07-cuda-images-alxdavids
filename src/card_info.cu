#include <container_ops.h>


int main(int argc, char **argv) {
    int deviceCount;

    hipGetDeviceCount(&deviceCount);

    for (int dev = 0; dev < deviceCount; dev++) {
        hipDeviceProp_t deviceProp;

        hipGetDeviceProperties(&deviceProp, dev);

        if (dev == 0) {
            if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
                LOG("No CUDA GPU has been detected");
                return -1;
            } else if (deviceCount == 1) {

                LOG("There is 1 device supporting CUDA");
            } else {
                LOG("There are " << deviceCount << " devices supporting CUDA");
            }
        }

        LOG("Device " << dev << " name: " << deviceProp.name);
        LOG(" Computational Capabilities: " << deviceProp.major << "." << deviceProp.minor);
        LOG(" Maximum global memory size: " << deviceProp.totalGlobalMem);
        LOG(" Maximum constant memory size: " << deviceProp.totalConstMem);
        LOG(" Maximum shared memory size per block: " << deviceProp.sharedMemPerBlock);
        LOG(" Maximum block dimensions: " << deviceProp.maxThreadsDim[0]
                 << " x " << deviceProp.maxThreadsDim[1]
                 << " x " << deviceProp.maxThreadsDim[2]);
        LOG(" Maximum grid dimensions: " << deviceProp.maxGridSize[0]
                 << " x " << deviceProp.maxGridSize[1]
                 << " x " << deviceProp.maxGridSize[2]);
        LOG(" Warp size: " << deviceProp.warpSize);

        return 0;
    }

    return 0;
}

