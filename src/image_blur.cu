#include "hip/hip_runtime.h"
#include <wb.h>


#define BLUR_SIZE 5 // The halo of the Stencil computation

#define TILE_WIDTH 16

/**
 * Blur filter
 *
 * @param out The buffer to receive the blurred image to compute
 * @param in The image to blur
 * @param width Width of in
 * @param height Height of in
 */
__global__ void blur_kernel(float *out, float *in, int width, int height) {
    // TODO
}




wbImage_t image_blur(wbImage_t inputImage) {
    const int imageWidth = wbImage_getWidth(inputImage);
    const int imageHeight = wbImage_getHeight(inputImage);

    const int imageChannels = 1; // Assuming grayscale images
    float *hostInputImageData = wbImage_getData(inputImage);

    // Since the image is monochromatic, it only contains one channel
    wbImage_t outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);
    float *hostOutputImageData = wbImage_getData(outputImage);

    // TODO

    // declare buffers for the images in the device

    // Allocate memory for the buffers in the device.
    // The size of each buffer is imageWidth * imageHeight * sizeof(float))

    // Copy the source image to the device

    // Define a 2-dimensional thread grid of tiles TILE_WIDTH x TILE_WIDTH

    // Execute the kernel

    // Copy the result to the main memory

    // Free the buffers on the device

    return outputImage;
}

int main(int argc, char *argv[]) {

    if (argc != 2) {
        printf ("usage %s image_file.ppm\n.", argv[0]);
        return 1;
    }
    const char *inputImageFile = argv[1];

    const wbImage_t inputImage = wbImport(inputImageFile);
    wbImage_t outputImage = image_blur(inputImage);
    wbExport("output.ppm", outputImage);

    wbImage_delete(outputImage);
    wbImage_delete(inputImage);

    return 0;
}