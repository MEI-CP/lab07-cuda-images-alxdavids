#include "hip/hip_runtime.h"
#include <wb.h>


#define TILE_WIDTH 16

/**
 * Kernel that converts a RGB image to gray image
 * @param grayImage The buffer to receive the grayscale image being computed
 * @param rgbImage The RGB image to convert
 * @param width Width of rgbImage
 * @param height Height of rgbImage
 */
__global__ void rgb2gray_cuda(float *grayImage, float *rgbImage, int width, int height) {

    ////////////
    // TODO
    //
    // Suggestion: obtain ii and jj from the lab sheet's pseudo code
    //             by using threadIdx.{x,y}, blockIdx.{x,y} and blockDim..{x,y}
    ////////////

}

wbImage_t rgb2gray(wbImage_t inputImage) {
    const int imageWidth = wbImage_getWidth(inputImage);
    const int imageHeight = wbImage_getHeight(inputImage);

    const int imageChannels = wbImage_getChannels(inputImage);   // For this lab the value is always 3
    float *hostInputImageData = wbImage_getData(inputImage);

    // Since the image is monochromatic, it only contains one channel
    wbImage_t outputImage = wbImage_new(imageWidth, imageHeight, 1);
    float *hostOutputImageData = wbImage_getData(outputImage);


    // TODO

    // declare buffers for the images in the device

    // Allocate memory for the buffers in the device.
    // The size of each buffer is imageWidth * imageHeight * imageChannels * sizeof(float))
    // where imageChannels is 3 for the RGB image and 1 for the grayscale image

    // Copy the source image to the device

    // Define a 2-dimensional thread grid of tiles TILE_WIDTH x TILE_WIDTH

    // Execute the kernel

    // Copy the result to the main memory

    // Free the buffers on the device

    return outputImage;
}

int main(int argc, char *argv[]) {

    if (argc != 2) {
        printf ("usage %s image_file.ppm\n.", argv[0]);
        return 1;
    }
    const char *inputImageFile = argv[1];

    const wbImage_t inputImage = wbImport(inputImageFile);
    wbImage_t outputImage = rgb2gray(inputImage);
    wbExport("output.ppm", outputImage);

    wbImage_delete(outputImage);
    wbImage_delete(inputImage);

    return 0;
}